#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (C) 2011-2014, NVIDIA Corporation
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 2 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

// alignment_test.cu
//

#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/dna.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/io/sequence/sequence_mmap.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

namespace nvbio {

int sequence_test(int argc, char* argv[])
{
    char* index_name = NULL;
    char* reads_name = NULL;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-map" ) == 0)
            index_name = argv[++i];
        else if (strcmp( argv[i], "-reads" ) == 0)
            reads_name = argv[++i];
    }

    log_info(stderr,"testing sequence-data... started\n");

    try
    {
        if (index_name != NULL)
        {
            log_verbose(stderr, "  loading sequence file %s\n", index_name );

            // try to load the index in memory
            io::SequenceDataHost index;
            if (io::load_sequence_file(
                DNA,
                &index,
                index_name ) == false)
            {
                log_error(stderr,"  loading file %s failed\n", index_name);
                return 0;
            }

            log_verbose(stderr, "  sequences : %u\n", index.size() );
            log_verbose(stderr, "  bps       : %u\n", index.bps() );
            log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                index.avg_sequence_len(),
                index.min_sequence_len(),
                index.max_sequence_len() );

            // try to load the index in mapped-memory
            io::SequenceDataMMAPServer server;
            if (server.load( DNA, index_name, "test", io::SequenceFlags( io::SEQUENCE_DATA | io::SEQUENCE_NAMES ) ) == false)
            {
                log_error(stderr,"  server mapping of file %s failed\n", index_name);
                return 0;
            }

            // scope the client so as to make sure it's destroyed before the server
            {
                // and map it into a client
                io::SequenceDataMMAP client;
                if (client.load( "test" ) == false)
                {
                    log_error(stderr,"  client mapping of file %s failed\n", index_name);
                    return 0;
                }

                log_verbose(stderr, "  sequences : %u\n", client.size() );
                log_verbose(stderr, "  bps       : %u\n", client.bps() );
                log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                    client.avg_sequence_len(),
                    client.min_sequence_len(),
                    client.max_sequence_len() );

                // check whether the stats match
                if (static_cast<const io::SequenceDataInfo&>( index ) !=
                    static_cast<const io::SequenceDataInfo&>( client ))
                {
                    log_error(stderr,"  loaded and mapped versions of file %s do not match!\n", index_name);
                    return 0;
                }
            }
        }
        if (reads_name != NULL)
        {
            SharedPointer<io::SequenceDataStream> read_file( io::open_sequence_file( reads_name ) );
            if (read_file == NULL || read_file->is_ok() == false)
            {
                log_error(stderr,"  failed opening reads file %s\n", reads_name);
                return 0;
            }

            io::SequenceDataHost read_data;

            io::next( DNA_N, &read_data, read_file.get(), 10000 );

            log_verbose(stderr, "  sequences : %u\n", read_data.size() );
            log_verbose(stderr, "  bps       : %u\n", read_data.bps() );
            log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                read_data.avg_sequence_len(),
                read_data.min_sequence_len(),
                read_data.max_sequence_len() );
        }
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
        return 0;
    }

    log_info(stderr,"testing sequence-data... done\n");
    return 1;
}

} // namespace nvbio
