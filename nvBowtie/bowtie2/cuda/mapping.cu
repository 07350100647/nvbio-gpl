#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (C) 2012-2014, NVIDIA Corporation
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 2 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <nvBowtie/bowtie2/cuda/mapping.h>

namespace nvbio {
namespace bowtie2 {
namespace cuda {

//
// For all i in [0, #seed hit ranges[, output the seed hit range size in
// out_ranges[i].
//
__global__ 
void gather_ranges_kernel(
    const uint32                        count,
    const uint32                        n_reads,
    const SeedHitDequeArrayDeviceView   hits,
    const uint32*                       hit_counts_scan,
          uint64*                       out_ranges)
{
    const uint32 thread_id = threadIdx.x + BLOCKDIM*blockIdx.x;
    if (thread_id >= count) return;

    // do a binary search, looking for thread_id in hit_counts_scan,
    // to find the corresponding read id.
    const uint32 read_id = uint32( upper_bound( thread_id, hit_counts_scan, n_reads ) - hit_counts_scan );

    // at this point we can figure out which seed hit / SA range this thread is
    // responsible of
    const uint32 count_offset = read_id ? hit_counts_scan[read_id-1] : 0u;

    const uint32 range_id = thread_id - count_offset;

    const SeedHit* hits_data = hits.get_data( read_id );

    const uint2 range = hits_data[ range_id ].get_range();
 
    // and we can compute the corresponding range size
    out_ranges[ thread_id ] = range.y - range.x;
}

//
// dispatch the call to gather_ranges_kernel
//
void gather_ranges(
    const uint32                        count,
    const uint32                        n_reads,
    const SeedHitDequeArrayDeviceView   hits,
    const uint32*                       hit_counts_scan,
          uint64*                       out_ranges)
{
    const int blocks = (count + BLOCKDIM-1) / BLOCKDIM;

    gather_ranges_kernel<<<blocks, BLOCKDIM>>>( count, n_reads, hits, hit_counts_scan, out_ranges );
}


//
// perform exact read mapping
//
void map_whole_read(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params)
{
    map_whole_read_t( read_batch, fmi, rfmi, queues, hits, params );
}

//
// perform one run of exact seed mapping for all the reads in the input queue,
// writing reads that need another run in the output queue
//
void map_exact(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params)
{
    map_exact_t( read_batch, fmi, rfmi, retry, queues, hits, params );
}

//
// perform multiple runs of exact seed mapping in one go and keep the best
//
void map_exact(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    SeedHitDequeArrayDeviceView                     hits,
    const uint2                                     seed_range,
    const ParamsPOD                                 params)
{
    map_exact_t( read_batch, fmi, rfmi, hits, seed_range, params );
}

//
// perform one run of approximate seed mapping for all the reads in the input queue,
// writing reads that need another run in the output queue
//
void map_approx(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params)
{
    map_approx_t( read_batch, fmi, rfmi, retry, queues, hits, params );
}

//
// perform multiple runs of approximate seed mapping in one go and keep the best
//
void map_approx(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    SeedHitDequeArrayDeviceView                     hits,
    const uint2                                     seed_range,
    const ParamsPOD                                 params)
{
    map_approx_t( read_batch, fmi, rfmi, hits, seed_range, params );
}

//
// perform one run of seed mapping
//
void map(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params)
{
    map_t( read_batch, fmi, rfmi, retry, queues, hits, params );
}

} // namespace cuda
} // namespace bowtie2
} // namespace nvbio
